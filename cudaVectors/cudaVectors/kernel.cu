
#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <time.h>
#include <stdio.h>

__global__ void addition(int *c, int *a, int *b)
{
	int index = (blockIdx.x<<27)+(threadIdx.x<<18)+(threadIdx.y<<9)+threadIdx.z;
	c[index] = a[index] + b[index];
}

int main() {
	int arraySize;
	scanf("%d", &arraySize);
	clock_t start = clock();
	int *a;
	int *b;
	int *c;
	c = (int *)malloc(arraySize*sizeof(int));
	a = (int *)malloc(arraySize*sizeof(int));
	b = (int *)malloc(arraySize*sizeof(int));
	for (int i = 0; i < arraySize; i++) {
		c[i] = 0;
		a[i] = 99999;
		b[i] = 99999;
	}

	int *d_a=0;
	int *d_b=0;
	int *d_c=0;
	hipMalloc((void **)&d_a, arraySize*sizeof(int));
	hipMalloc((void **)&d_b, arraySize*sizeof(int));
	hipMalloc((void **)&d_c, arraySize*sizeof(int));

	hipMemcpy(d_a, a, arraySize*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, arraySize*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, arraySize*sizeof(int), hipMemcpyHostToDevice);

	//int blocks = (arraySize >> 27);
	addition <<<dim3(10,1,1), dim3(512,512,512)>>>(d_c, d_b, d_a);
	hipMemcpy(c, d_c, arraySize*sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	free(a);
	free(b);
	free(c);
	clock_t end = clock();
	float seconds = (float)(end - start) / CLOCKS_PER_SEC;
	printf("time cost: %f\n", seconds);
	/*
	for (int i = 0; i < arraySize;i++) {
		printf("%d ", c[i]);
	}*/
    return 0;
}
